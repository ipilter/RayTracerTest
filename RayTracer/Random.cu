#include "hip/hip_runtime.h"
#include "Random.cuh"

#include <hip/hip_runtime.h>
#include <>
#include <hiprand.h>

namespace random
{

__global__ void InitRandomStates( uint32_t seed
                            , math::uvec2 size
                            , hiprandState_t* states )
{
  const uint32_t x( blockIdx.x * blockDim.x + threadIdx.x );
  const uint32_t y( blockIdx.y * blockDim.y + threadIdx.y );
  if ( x >= size.x || y >= size.y )
  {
    return;
  }

  const size_t offset( x + y * size.x );
  hiprandState_t state( states[offset] );

  hiprand_init( seed,   // the seed can be the same for each core
               offset, // the sequence number should be different for each core (unless you want all cores to get the same sequence of numbers for some reason - use thread id!
               0,      // the offset is how much extra we advance in the sequence for each call, can be 0
               &state );

  states[offset] = state;
}

__host__ void CreateStates( const math::uvec2& size, hiprandState_t*& states )
{
  hipError_t err = hipMalloc( reinterpret_cast<void**>( &states ), size.x * size.y * sizeof( hiprandState_t ) );
  if ( err != hipSuccess )
  {
    throw std::runtime_error( std::string( "hipMalloc failed. (" ) + hipGetErrorString( err ) + ")\n");
  }

  const dim3 threadsPerBlock( 32, 32, 1 );
  const dim3 blocksPerGrid( static_cast<uint32_t>( glm::ceil( size.x / static_cast<float>( threadsPerBlock.x ) ) )
                            , static_cast<uint32_t>( glm::ceil( size.y / static_cast<float>( threadsPerBlock.y ) ) )
                            , 1 );

  InitRandomStates<<<blocksPerGrid, threadsPerBlock>>>( static_cast<uint32_t>( time( nullptr ) ), size, states );

  err = hipGetLastError();
  if ( err != hipSuccess )
  {
    throw std::runtime_error( std::string( "InitRandomKernel failed. (" ) + hipGetErrorString( err ) + ")\n");
  }
}

}
