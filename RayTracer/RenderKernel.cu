#include "hip/hip_runtime.h"
﻿
#include <sstream>
#include <iomanip>
#include <cstdio>
#include <cmath>

#include <hip/hip_runtime.h>
#include <>

#include "RenderData.cuh"

#include "Utils.cuh"
#include "ThinLensCamera.cuh"

// Note: arguments MUST be by value. Make sure they are fast to copy
__global__ void RenderKernel( rt::RenderData renderData )
{
  using namespace math;

  const uvec2 pixel( blockIdx.x * blockDim.x + threadIdx.x, blockIdx.y * blockDim.y + threadIdx.y );
  if ( pixel.x >= renderData.mDimensions.x || pixel.y >= renderData.mDimensions.y )
  {
    return;
  }

  const uint32_t offset( pixel.x + pixel.y * renderData.mDimensions.x );
  renderData.mRandom.SetOffset( offset );

  vec3 accu( 0.0f );
  for ( auto s = 0; s < renderData.mSampleCount; ++s )
  {
    const rt::Ray ray( renderData.mCamera.GetRay( pixel, renderData.mDimensions, renderData.mRandom ) );
    accu += ray.direction();
  }
  accu /= renderData.mSampleCount; // static_cast<float>( );

  // save final pixel color
  renderData.mPixelBuffer[offset] = utils::Color( 255 * accu.x
                                                  , 255 * accu.y
                                                  , 255 * accu.z );
}

hipError_t RunRenderKernel( rt::RenderData& renderData )
{
  // TODO fast way, do better!
  const dim3 threadsPerBlock( 32, 32, 1 );
  const dim3 blocksPerGrid( static_cast<uint32_t>( glm::ceil( renderData.mDimensions.x / static_cast<float>( threadsPerBlock.x ) ) )
                            , static_cast<uint32_t>( glm::ceil( renderData.mDimensions.y / static_cast<float>( threadsPerBlock.y ) ) )
                            , 1 );

  hipEvent_t start, stop;
  hipEventCreate( &start );
  hipEventCreate( &stop );

  hipEventRecord( start, 0 );
  RenderKernel<<<blocksPerGrid, threadsPerBlock>>> ( renderData );
  hipEventRecord( stop, 0 );
  hipEventSynchronize( stop );

  float time = 0.0f;
  hipEventElapsedTime( &time, start, stop );
  return hipGetLastError();
}
