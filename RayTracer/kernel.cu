#include "hip/hip_runtime.h"
﻿
#include <sstream>
#include <iomanip>
#include <cstdio>
#include <cmath>

#include <hip/hip_runtime.h>
#include <>

#include "RenderData.h"

#include "Utils.cuh"

__global__ void UpdateTextureKernel( uint32_t* rgba, int const width, int const height )
{
  const uint32_t x( blockIdx.x * blockDim.x + threadIdx.x );
  const uint32_t y( blockIdx.y * blockDim.y + threadIdx.y );
  if ( x >= width || y >= height )
  {
    return;
  }

  const uint32_t offset = x + y * width;
  rgba[offset] = utils::Color( 255 * ( x / static_cast<float>( width ) )
                               , 255 * ( y / static_cast<float>( height ) )
                               , 0 );
}

hipError_t RunUpdateTextureKernel( rt::RenderData& renderData )
{
  // TODO fast way, do better!
  dim3 threadsPerBlock( 32, 32, 1 );
  dim3 blocksPerGrid( static_cast<uint32_t>( glm::ceil( renderData.Dimensions().x / static_cast<float>( threadsPerBlock.x ) ) )
                      , static_cast<uint32_t>( glm::ceil( renderData.Dimensions().y / static_cast<float>( threadsPerBlock.y ) ) )
                      , 1 );

  hipEvent_t start, stop;
  hipEventCreate( &start );
  hipEventCreate( &stop );

  hipEventRecord( start, 0 );
  UpdateTextureKernel<<<blocksPerGrid, threadsPerBlock>>> ( renderData.PixelBuffer(), renderData.Dimensions().x, renderData.Dimensions().y );
  hipEventRecord( stop, 0 );
  hipEventSynchronize( stop );

  float time = 0.0f;
  hipEventElapsedTime( &time, start, stop );
  return hipGetLastError();
}
