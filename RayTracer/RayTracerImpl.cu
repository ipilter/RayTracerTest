#include "hip/hip_runtime.h"
#include <hip/hip_runtime.h>
#include <chrono>
#include <>

#include "RayTracerImpl.cuh"
#include "Random.cuh"
#include "DeviceUtils.cuh"
#include "Kernels.cuh"
#include "RenderBuffer.cuh"
#include "RaytracerCallback.h"

#include "Common\Logger.h"

namespace rt
{

RayTracerImpl::RayTracerImpl( const math::uvec2& imageSize
                              , const math::vec3& cameraPosition
                              , const math::vec2& cameraAngles
                              , const float fov
                              , const float focalLength
                              , const float aperture )
  : mBufferSize( imageSize )
  , mRenderBuffer( nullptr )
  , mSampleCountBuffer( nullptr )
  , mImageBuffer( nullptr )
  , mRandomStates( nullptr )
  , mCamera( new rt::ThinLensCamera( cameraPosition, cameraAngles, fov, focalLength, aperture ) )
  , mStopped( false )
{
  try
  {
    random::CreateStates( mBufferSize, mRandomStates );
    rt::CreateRenderBuffer( mBufferSize, ChannelCount(), mRenderBuffer );
    rt::CreateSampleCountBuffer( mBufferSize, mSampleCountBuffer );
    rt::CreateImageBuffer( mBufferSize, mImageBuffer );

    logger::Logger::Instance() << "Raytracer created. Image buffer size: " << mBufferSize << "\n";
  }
  catch ( const std::exception& e )
  {
    logger::Logger::Instance() << "RayTracerImpl construction failed. Reason: " << e.what() << "\n";
  }
}

RayTracerImpl::~RayTracerImpl()
{
  // If rendering task is in progress, cancel it
  mStopped = true;
  if ( mThread.joinable() )
  {
    mThread.join();
  }

  // TODO free temporary image buffer - storage of converted image pixels (RGBA)
  ReleaseBuffers();
}

void RayTracerImpl::Trace( const uint32_t iterationCount
                           , const uint32_t samplesPerIteration
                           , const uint32_t updateInterval )
{
  // cancel previous operation, if any
  if ( mThread.joinable() )
  {
    mStopped = true;
    mThread.join();
    mStopped = false;
  }

  // Run rendering function async. TODO use pool instead of creating a new thread
  mThread = std::thread( std::bind( &RayTracerImpl::TraceFunct
                                    , this
                                    , iterationCount
                                    , samplesPerIteration
                                    , updateInterval ) );
}

void RayTracerImpl::Stop()
{
  mStopped = true;
}

void RayTracerImpl::Resize( const math::uvec2& size )
{
  ReleaseBuffers();

  mBufferSize = size;
  random::CreateStates( mBufferSize, mRandomStates );
  rt::CreateRenderBuffer( mBufferSize, ChannelCount(), mRenderBuffer );
  rt::CreateSampleCountBuffer( mBufferSize, mSampleCountBuffer );
  rt::CreateImageBuffer( mBufferSize, mImageBuffer );
}

void RayTracerImpl::SetCameraParameters( const float fov
                                         , const float focalLength
                                         , const float aperture )
{
  mCamera->Fov( fov );
  mCamera->FocalLength( focalLength );
  mCamera->Aperture( aperture );
}

void RayTracerImpl::RotateCamera( const math::vec2& angles )
{
  mCamera->Rotate( angles );
}

void RayTracerImpl::SetUpdateCallback( rt::CallBackFunction callback )
{
  mUpdateCallback = callback;
}

void RayTracerImpl::SetFinishedCallback( rt::CallBackFunction callback )
{
  mFinishedCallback = callback;
}

hipError_t RayTracerImpl::RunConverterKernel()
{
  const dim3 threadsPerBlock( 32, 32, 1 );
  const dim3 blocksPerGrid( static_cast<uint32_t>( glm::ceil( mBufferSize.x / static_cast<float>( threadsPerBlock.x ) ) )
                            , static_cast<uint32_t>( glm::ceil( mBufferSize.y / static_cast<float>( threadsPerBlock.y ) ) )
                            , 1 );

  ConverterKernel<<<blocksPerGrid, threadsPerBlock>>>( mBufferSize
                                                       , ChannelCount()
                                                       , mRenderBuffer
                                                       , mSampleCountBuffer
                                                       , mImageBuffer );
  return hipGetLastError();
}

hipError_t RayTracerImpl::RunTraceKernel( const uint32_t sampleCount )
{
  // TODO fast way, do better!
  const dim3 threadsPerBlock( 32, 32, 1 );
  const dim3 blocksPerGrid( static_cast<uint32_t>( glm::ceil( mBufferSize.x / static_cast<float>( threadsPerBlock.x ) ) )
                            , static_cast<uint32_t>( glm::ceil( mBufferSize.y / static_cast<float>( threadsPerBlock.y ) ) )
                            , 1 );

  hipEvent_t start, stop;
  hipEventCreate( &start );
  hipEventCreate( &stop );

  hipEventRecord( start, 0 );
  TraceKernel<<<blocksPerGrid, threadsPerBlock>>> ( mRenderBuffer
                                                    , mSampleCountBuffer
                                                    , mBufferSize
                                                    , ChannelCount()
                                                    , *mCamera
                                                    , sampleCount
                                                    , mRandomStates );

  hipEventRecord( stop, 0 );
  hipEventSynchronize( stop ); // TODO: make this switchable (on/off)

  float time = 0.0f;
  hipEventElapsedTime( &time, start, stop );

  return hipGetLastError();
}

__host__ void RayTracerImpl::TraceFunct( const uint32_t iterationCount
                                         , const uint32_t samplesPerIteration
                                         , const uint32_t updateInterval )
{
  try
  {
    rt::ClearRenderBuffer( mBufferSize, ChannelCount(), mRenderBuffer );
    rt::ClearSampleCountBuffer( mBufferSize, mSampleCountBuffer );

    hipError_t err = hipSuccess;
    for ( uint32_t i( 0 ); !mStopped && i < iterationCount; ++i )
    {
      // TODO: make kernel call cancellable if possible (imageine long runtimer here, cancel operation would wait for this call)
      err = RunTraceKernel( samplesPerIteration );
      if ( err != hipSuccess )
      {
        throw std::runtime_error( std::string( "RunTraceKernel failed: " ) + hipGetErrorString( err ) );
      }

      // check if update is needed
      if ( mUpdateCallback != nullptr && i > 0 && updateInterval > 0 && i % updateInterval == 0 )
      {
        // wait for the scheduled commands to be executed
        hipDeviceSynchronize();

        // run render -> image conversion
        err = RunConverterKernel();
        if ( err != hipSuccess )
        {
          throw std::runtime_error( std::string( "RunConverterKernel failed: " ) + hipGetErrorString( err ) );
        }

        // notify view to update the view's texture
        mUpdateCallback( mImageBuffer, mBufferSize.x * mBufferSize.y * sizeof( rt::Color ) );
      }

      using namespace std::chrono_literals;
      //std::this_thread::sleep_for( 500ms );
    }

    // early reaturn if cancel was called on us
    if ( mStopped )
    {
      // TODO: any UI update?
      return;
    }

    // wait for the scheduled commands to be executed
    hipDeviceSynchronize();

    // run render -> image conversion
    err = RunConverterKernel();
    if ( err != hipSuccess )
    {
      throw std::runtime_error( std::string( "RunConverterKernel failed: " ) + hipGetErrorString( err ) );
    }

    // notify view that we are done
    if ( mFinishedCallback != nullptr )
    {
      mFinishedCallback( mImageBuffer, mBufferSize.x * mBufferSize.y * sizeof( rt::Color ) );
    }
  }
  catch ( const std::exception& /*e*/ )
  {
    // TODO: error handling
  }
  catch ( ... )
  {
    // TODO: error handling
  }
}

void RayTracerImpl::ReleaseBuffers()
{
  hipError_t err = hipFree( mRandomStates );
  if ( err != hipSuccess )
  {
    logger::Logger::Instance() << "Error: hipFree failed freeing mRandomStates. (" << hipGetErrorString( err ) << "\n";
  }

  err = hipFree( mRenderBuffer );
  if ( err != hipSuccess )
  {
    logger::Logger::Instance() << "Error: hipFree failed freeing mRenderBuffer. (" << hipGetErrorString( err ) << "\n";
  }

  err = hipFree( mSampleCountBuffer );
  if ( err != hipSuccess )
  {
    logger::Logger::Instance() << "Error: hipFree failed freeing mSampleCountBuffer. (" << hipGetErrorString( err ) << "\n";
  }

  err = hipFree( mImageBuffer );
  if ( err != hipSuccess )
  {
    logger::Logger::Instance() << "Error: hipFree failed freeing mImageBuffer. (" << hipGetErrorString( err ) << "\n";
  }
}

}
