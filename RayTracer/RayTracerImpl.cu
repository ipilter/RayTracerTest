#include "hip/hip_runtime.h"
#include <hip/hip_runtime.h>
#include <>

#include "RayTracerImpl.cuh"
#include "Random.cuh"
#include "DeviceUtils.cuh"
#include "RenderKernel.cuh"

#include "Common\Logger.h"

namespace rt
{

RayTracerImpl::RayTracerImpl( const math::uvec2& pixelBufferSize
                              , const float fov
                              , const float focalLength
                              , const float aperture )
  : mPixelBufferSize( pixelBufferSize )
  , mRandomStates( nullptr )
  , mCamera( new rt::ThinLensCamera( math::vec3( 0.0f, 0.0f, 0.0f )
                                     , math::vec3( 0.0f, 0.0f, -1.0f )
                                     , math::vec3( 0.0f, 1.0f, 0.0f )
                                     , fov, focalLength, aperture ) )
{
  logger::Logger::Instance() << "Raytracer created. Pixel buffer size: " << pixelBufferSize << "\n";
  random::RunInitRandomKernel( pixelBufferSize, mRandomStates );
}

RayTracerImpl::~RayTracerImpl()
{
  const hipError_t err = hipFree( mRandomStates );
  if ( err != hipSuccess )
  {
    logger::Logger::Instance() << "Error: hipFree failed freeing mRandomStates. (" << hipGetErrorString( err ) << "\n";
  }
}

void RayTracerImpl::Trace( rt::color_t* pixelBufferPtr, const uint32_t sampleCount )
{
  hipError_t err = RunRenderKernel( pixelBufferPtr, mPixelBufferSize, *mCamera, sampleCount, mRandomStates );
  if ( err != hipSuccess )
  {
    throw std::runtime_error( std::string( "RunRenderKernel failed: " ) + hipGetErrorString( err ) );
  }
}

void RayTracerImpl::Resize( const math::uvec2& size )
{
  mPixelBufferSize = size;
  const hipError_t err = hipFree( mRandomStates );
  if ( err != hipSuccess )
  {
    throw std::runtime_error( std::string( "hipFree failed: " ) + hipGetErrorString( err ) );
  }

  random::RunInitRandomKernel( size, mRandomStates );
}

void RayTracerImpl::SetCameraParameters( const float fov
                                         , const float focalLength
                                         , const float aperture )
{
  mCamera->Fov( fov );
  mCamera->FocalLength( focalLength );
  mCamera->Aperture( aperture );
}

void RayTracerImpl::RotateCamera( const math::uvec2& angles )
{
  mCamera->Rotate( angles );
}

hipError_t RayTracerImpl::RunRenderKernel( rt::color_t* pixelBufferPtr
                                            , const math::uvec2& pixelBufferSize
                                            , rt::ThinLensCamera& camera
                                            , const uint32_t sampleCount
                                            , hiprandState_t* randomStates )
{
  // TODO fast way, do better!
  const dim3 threadsPerBlock( 32, 32, 1 );
  const dim3 blocksPerGrid( static_cast<uint32_t>( glm::ceil( pixelBufferSize.x / static_cast<float>( threadsPerBlock.x ) ) )
                            , static_cast<uint32_t>( glm::ceil( pixelBufferSize.y / static_cast<float>( threadsPerBlock.y ) ) )
                            , 1 );

  hipEvent_t start, stop;
  hipEventCreate( &start );
  hipEventCreate( &stop );

  hipEventRecord( start, 0 );
  RenderKernel<<<blocksPerGrid, threadsPerBlock>>> ( pixelBufferPtr, pixelBufferSize, camera, sampleCount, randomStates );

  // this blocks the CPU till all the GPU commands are executed (kernel, copy, etc)
  //hipDeviceSynchronize();

  hipEventRecord( stop, 0 );
  hipEventSynchronize( stop ); // TODO: make this switchable (on/off)

  float time = 0.0f;
  hipEventElapsedTime( &time, start, stop );
  logger::Logger::Instance() << "Render kernel runtime: " << time << " ms\n";

  return hipGetLastError();
}

}
