#include "hip/hip_runtime.h"
#include <hip/hip_runtime.h>
#include <>

#include "RayTracerImpl.cuh"
#include "Random.cuh"
#include "DeviceUtils.cuh"
#include "RenderKernel.cuh"
#include "Render.cuh"

#include "Common\Logger.h"

namespace rt
{

RayTracerImpl::RayTracerImpl( const math::uvec2& pixelBufferSize
                              , const math::vec3& cameraPosition
                              , const math::vec2& cameraAngles
                              , const float fov
                              , const float focalLength
                              , const float aperture )
  : mPixelBufferSize( pixelBufferSize )
  , mRandomStates( nullptr )
  , mRenderBuffer( nullptr )
  , mCamera( new rt::ThinLensCamera( cameraPosition, cameraAngles, fov, focalLength, aperture ) )
  , mCancelled( false )
  , mThread()
{
  const uint32_t channelCount = 4;

  random::CreateStates( mPixelBufferSize, mRandomStates );
  render::CreateRenderBuffer( mPixelBufferSize, channelCount, mRenderBuffer );
  render::ClearRenderBuffer( mPixelBufferSize, channelCount, mRenderBuffer );

  logger::Logger::Instance() << "Raytracer created. Pixel buffer size: " << mPixelBufferSize << "\n";
}

RayTracerImpl::~RayTracerImpl()
{
  // If rendering task is in progress, cancel it
  mCancelled = true;
  if ( mThread.joinable() )
  {
    mThread.join();
  }

  hipError_t err = hipFree( mRandomStates );
  if ( err != hipSuccess )
  {
    logger::Logger::Instance() << "Error: hipFree failed freeing mRandomStates. (" << hipGetErrorString( err ) << "\n";
  }

  err = hipFree( mRenderBuffer );
  if ( err != hipSuccess )
  {
    logger::Logger::Instance() << "Error: hipFree failed freeing mRenderBuffer. (" << hipGetErrorString( err ) << "\n";
  }
}

void RayTracerImpl::Trace( rt::color_t* pixelBufferPtr
                           , const uint32_t iterationCount
                           , const uint32_t samplesPerIteration
                           , const uint32_t updatesOnIteration )
{
  // cancel previous operation, if any
  mCancelled = true;
  if ( mThread.joinable() )
  {
    mThread.join();
  }
  mCancelled = false;

  // Run rendering function async
  mThread = std::thread( std::bind( &RayTracerImpl::TraceFunct
                                    , this
                                    , pixelBufferPtr
                                    , iterationCount
                                    , samplesPerIteration
                                    , updatesOnIteration ) );
}

void RayTracerImpl::Cancel()
{
  mCancelled = true;
}

void RayTracerImpl::Resize( const math::uvec2& size )
{
  mPixelBufferSize = size;
  hipError_t err = hipFree( mRandomStates );
  if ( err != hipSuccess )
  {
    logger::Logger::Instance() << "Error: hipFree failed freeing mRandomStates. (" << hipGetErrorString( err ) << "\n";
  }

  err = hipFree( mRenderBuffer );
  if ( err != hipSuccess )
  {
    logger::Logger::Instance() << "Error: hipFree failed freeing mRenderBuffer. (" << hipGetErrorString( err ) << "\n";
  }

  random::CreateStates( mPixelBufferSize, mRandomStates );
  render::CreateRenderBuffer( mPixelBufferSize, 4, mRenderBuffer );
}

void RayTracerImpl::SetCameraParameters( const float fov
                                         , const float focalLength
                                         , const float aperture )
{
  mCamera->Fov( fov );
  mCamera->FocalLength( focalLength );
  mCamera->Aperture( aperture );
}

void RayTracerImpl::RotateCamera( const math::vec2& angles )
{
  mCamera->Rotate( angles );
}

void RayTracerImpl::SetUpdateCallback( CallBackFunction callback )
{
  mUpdateCallback = callback;
}

void RayTracerImpl::SetFinishedCallback( CallBackFunction callback )
{
  mFinishedCallback = callback;
}

hipError_t RayTracerImpl::RunConverterKernel( const math::uvec2& bufferSize
                                                      , const uint32_t channelCount
                                                      , float*& renderBuffer
                                                      , rt::color_t* pixelBufferPtr )
{
  const dim3 threadsPerBlock( 32, 32, 1 );
  const dim3 blocksPerGrid( static_cast<uint32_t>( glm::ceil( bufferSize.x / static_cast<float>( threadsPerBlock.x ) ) )
                            , static_cast<uint32_t>( glm::ceil( bufferSize.y / static_cast<float>( threadsPerBlock.y ) ) )
                            , 1 );

  ConverterKernel<<<blocksPerGrid, threadsPerBlock>>>( bufferSize, channelCount, renderBuffer, pixelBufferPtr );
  return hipGetLastError();
}

hipError_t RayTracerImpl::RunRenderKernel( float* renderBuffer
                                            , const math::uvec2& bufferSize
                                            , const uint32_t channelCount
                                            , rt::ThinLensCamera& camera
                                            , const uint32_t sampleCount
                                            , hiprandState_t* randomStates )
{
  // TODO fast way, do better!
  const dim3 threadsPerBlock( 32, 32, 1 );
  const dim3 blocksPerGrid( static_cast<uint32_t>( glm::ceil( bufferSize.x / static_cast<float>( threadsPerBlock.x ) ) )
                            , static_cast<uint32_t>( glm::ceil( bufferSize.y / static_cast<float>( threadsPerBlock.y ) ) )
                            , 1 );

  hipEvent_t start, stop;
  hipEventCreate( &start );
  hipEventCreate( &stop );

  hipEventRecord( start, 0 );
  RenderKernel<<<blocksPerGrid, threadsPerBlock>>> ( renderBuffer, bufferSize, channelCount, camera, sampleCount, randomStates );

  hipEventRecord( stop, 0 );
  hipEventSynchronize( stop ); // TODO: make this switchable (on/off)

  //hipDeviceSynchronize(); // this blocks the CPU till all the GPU commands are executed (kernel, copy, etc)

  float time = 0.0f;
  hipEventElapsedTime( &time, start, stop );

  return hipGetLastError();
}

__host__ void RayTracerImpl::TraceFunct( rt::color_t* pixelBufferPtr
                                         , const uint32_t iterationCount
                                         , const uint32_t samplesPerIteration
                                         , const uint32_t updatesOnIteration )
{
  try
  {
    const uint32_t channelCount = 4;

    render::ClearRenderBuffer( mPixelBufferSize, channelCount, mRenderBuffer );

    hipError_t err = hipSuccess;
    for ( auto i = 0u; i < iterationCount; ++i ) // TODO && !mIsCancelled
    {
      RunRenderKernel( mRenderBuffer, mPixelBufferSize, channelCount, *mCamera, samplesPerIteration, mRandomStates );
      if ( err != hipSuccess )
      {
        throw std::runtime_error( std::string( "RunRenderKernel failed: " ) + hipGetErrorString( err ) );
      }

      if ( mUpdateCallback != nullptr && updatesOnIteration > 0 && i % updatesOnIteration == 0 )
      {
        err = RunConverterKernel( mPixelBufferSize, channelCount, mRenderBuffer, pixelBufferPtr );
        if ( err != hipSuccess )
        {
          throw std::runtime_error( std::string( "RunConverterKernel failed: " ) + hipGetErrorString( err ) );
        }

        // notify view to update the view's texture
        mUpdateCallback();
      }
    }

    // convert render buffer to image and store it in PBO
    err = RunConverterKernel( mPixelBufferSize, channelCount, mRenderBuffer, pixelBufferPtr );
    if ( err != hipSuccess )
    {
      throw std::runtime_error( std::string( "RunConverterKernel failed: " ) + hipGetErrorString( err ) );
    }

    // notify view that we are done
    if ( mFinishedCallback != nullptr )
    {
      mFinishedCallback();
    }
  }
  catch ( const std::exception& e )
  {
    // TODO: error handling
  }
}

}
