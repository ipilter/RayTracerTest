#include "hip/hip_runtime.h"
#include <hip/hip_runtime.h>
#include <>

#include "RayTracerImpl.cuh"
#include "Random.cuh"
#include "DeviceUtils.cuh"
#include "Kernels.cuh"
#include "RenderBuffer.cuh"

#include "Common\Logger.h"

namespace rt
{

RayTracerImpl::RayTracerImpl( const math::uvec2& pixelBufferSize
                              , const math::vec3& cameraPosition
                              , const math::vec2& cameraAngles
                              , const float fov
                              , const float focalLength
                              , const float aperture )
  : mPixelBufferSize( pixelBufferSize )
  , mRandomStates( nullptr )
  , mRenderBuffer( nullptr )
  , mCamera( new rt::ThinLensCamera( cameraPosition, cameraAngles, fov, focalLength, aperture ) )
  , mCancelled( false )
  , mThread()
{
  const uint32_t channelCount = 4;

  random::CreateStates( mPixelBufferSize, mRandomStates );
  render::CreateRenderBuffer( mPixelBufferSize, channelCount, mRenderBuffer );
  render::ClearRenderBuffer( mPixelBufferSize, channelCount, mRenderBuffer );

  logger::Logger::Instance() << "Raytracer created. Pixel buffer size: " << mPixelBufferSize << "\n";
}

RayTracerImpl::~RayTracerImpl()
{
  // If rendering task is in progress, cancel it
  mCancelled = true;
  if ( mThread.joinable() )
  {
    mThread.join();
  }

  hipError_t err = hipFree( mRandomStates );
  if ( err != hipSuccess )
  {
    logger::Logger::Instance() << "Error: hipFree failed freeing mRandomStates. (" << hipGetErrorString( err ) << "\n";
  }

  err = hipFree( mRenderBuffer );
  if ( err != hipSuccess )
  {
    logger::Logger::Instance() << "Error: hipFree failed freeing mRenderBuffer. (" << hipGetErrorString( err ) << "\n";
  }
}

void RayTracerImpl::Trace( hipGraphicsResource_t pboCudaResource
                           , const uint32_t iterationCount
                           , const uint32_t samplesPerIteration
                           , const uint32_t updatesOnIteration )
{
  // cancel previous operation, if any
  if ( mThread.joinable() )
  {
    mCancelled = true;
    mThread.join();
    mCancelled = false;
  }

  // Run rendering function async. TODO use pool instead of creating a new thread
  mThread = std::thread( std::bind( &RayTracerImpl::TraceFunct
                                    , this
                                    , pboCudaResource
                                    , iterationCount
                                    , samplesPerIteration
                                    , updatesOnIteration ) );
}

void RayTracerImpl::Cancel()
{
  mCancelled = true;
}

void RayTracerImpl::Resize( const math::uvec2& size )
{
  mPixelBufferSize = size;
  hipError_t err = hipFree( mRandomStates );
  if ( err != hipSuccess )
  {
    logger::Logger::Instance() << "Error: hipFree failed freeing mRandomStates. (" << hipGetErrorString( err ) << "\n";
  }

  err = hipFree( mRenderBuffer );
  if ( err != hipSuccess )
  {
    logger::Logger::Instance() << "Error: hipFree failed freeing mRenderBuffer. (" << hipGetErrorString( err ) << "\n";
  }

  random::CreateStates( mPixelBufferSize, mRandomStates );
  render::CreateRenderBuffer( mPixelBufferSize, 4, mRenderBuffer );
}

void RayTracerImpl::SetCameraParameters( const float fov
                                         , const float focalLength
                                         , const float aperture )
{
  mCamera->Fov( fov );
  mCamera->FocalLength( focalLength );
  mCamera->Aperture( aperture );
}

void RayTracerImpl::RotateCamera( const math::vec2& angles )
{
  mCamera->Rotate( angles );
}

void RayTracerImpl::SetUpdateCallback( CallBackFunction callback )
{
  mUpdateCallback = callback;
}

void RayTracerImpl::SetFinishedCallback( CallBackFunction callback )
{
  mFinishedCallback = callback;
}

hipError_t RayTracerImpl::RunConverterKernel( const math::uvec2& bufferSize
                                                      , const uint32_t channelCount
                                                      , float*& renderBuffer
                                                      , rt::color_t* pixelBufferPtr )
{
  const dim3 threadsPerBlock( 32, 32, 1 );
  const dim3 blocksPerGrid( static_cast<uint32_t>( glm::ceil( bufferSize.x / static_cast<float>( threadsPerBlock.x ) ) )
                            , static_cast<uint32_t>( glm::ceil( bufferSize.y / static_cast<float>( threadsPerBlock.y ) ) )
                            , 1 );

  ConverterKernel<<<blocksPerGrid, threadsPerBlock>>>( bufferSize, channelCount, renderBuffer, pixelBufferPtr );
  return hipGetLastError();
}

hipError_t RayTracerImpl::RunTraceKernel( float* renderBuffer
                                           , const math::uvec2& bufferSize
                                           , const uint32_t channelCount
                                           , rt::ThinLensCamera& camera
                                           , const uint32_t sampleCount
                                           , hiprandState_t* randomStates )
{
  // TODO fast way, do better!
  const dim3 threadsPerBlock( 32, 32, 1 );
  const dim3 blocksPerGrid( static_cast<uint32_t>( glm::ceil( bufferSize.x / static_cast<float>( threadsPerBlock.x ) ) )
                            , static_cast<uint32_t>( glm::ceil( bufferSize.y / static_cast<float>( threadsPerBlock.y ) ) )
                            , 1 );

  hipEvent_t start, stop;
  hipEventCreate( &start );
  hipEventCreate( &stop );

  hipEventRecord( start, 0 );
  TraceKernel<<<blocksPerGrid, threadsPerBlock>>> ( mRenderBuffer, bufferSize, channelCount, camera, sampleCount, randomStates );

  hipEventRecord( stop, 0 );
  hipEventSynchronize( stop ); // TODO: make this switchable (on/off)

  //hipDeviceSynchronize(); // this blocks the CPU till all the GPU commands are executed (kernel, copy, etc)

  float time = 0.0f;
  hipEventElapsedTime( &time, start, stop );

  return hipGetLastError();
}

__host__ void RayTracerImpl::TraceFunct( hipGraphicsResource_t pboCudaResource
                                         , const uint32_t iterationCount
                                         , const uint32_t samplesPerIteration
                                         , const uint32_t updatesOnIteration )
{
  try
  {
    const uint32_t channelCount = 4;

    render::ClearRenderBuffer( mPixelBufferSize, channelCount, mRenderBuffer );

    hipError_t err = hipSuccess;
    for ( uint32_t i( 0 ); !mCancelled && i < iterationCount; ++i )
    {
      err = RunTraceKernel( mRenderBuffer, mPixelBufferSize, channelCount, *mCamera, samplesPerIteration, mRandomStates );
      if ( err != hipSuccess )
      {
        throw std::runtime_error( std::string( "RunTraceKernel failed: " ) + hipGetErrorString( err ) );
      }

      // check if update is needed
      if ( mUpdateCallback != nullptr && i > 0 && updatesOnIteration > 0 && i % updatesOnIteration == 0 )
      {
        err = hipGraphicsMapResources( 1, &pboCudaResource );
        if ( err != hipSuccess )
        {
          throw std::runtime_error( std::string( "hipGraphicsMapResources failed: " ) + hipGetErrorString( err ) );
        }

        rt::color_t* pixelBufferPtr = nullptr;
        size_t size = 0;
        err = hipGraphicsResourceGetMappedPointer( reinterpret_cast<void**>( &pixelBufferPtr )
                                                                , &size
                                                                , pboCudaResource );
        if ( err != hipSuccess )
        {
          throw std::runtime_error( std::string( "hipGraphicsResourceGetMappedPointer failed: " ) + hipGetErrorString( err ) );
        }


        err = RunConverterKernel( mPixelBufferSize, channelCount, mRenderBuffer, pixelBufferPtr );
        if ( err != hipSuccess )
        {
          throw std::runtime_error( std::string( "RunConverterKernel failed: " ) + hipGetErrorString( err ) );
        }

        err = hipGraphicsUnmapResources( 1, &pboCudaResource );
        if ( err != hipSuccess )
        {
          throw std::runtime_error( std::string( "hipGraphicsUnmapResources failed: " ) + hipGetErrorString( err ) );
        }

        // notify view to update the view's texture
        mUpdateCallback();
      }
    }

    // early reaturn if cancel was called on us
    if ( mCancelled )
    {
      return;
    }

    err = hipGraphicsMapResources( 1, &pboCudaResource );
    if ( err != hipSuccess )
    {
      throw std::runtime_error( std::string( "hipGraphicsMapResources failed: " ) + hipGetErrorString( err ) );
    }

    rt::color_t* pixelBufferPtr = nullptr;
    size_t size = 0;
    err = hipGraphicsResourceGetMappedPointer( reinterpret_cast<void**>( &pixelBufferPtr )
                                                , &size
                                                , pboCudaResource );
    if ( err != hipSuccess )
    {
      throw std::runtime_error( std::string( "hipGraphicsResourceGetMappedPointer failed: " ) + hipGetErrorString( err ) );
    }


    err = RunConverterKernel( mPixelBufferSize, channelCount, mRenderBuffer, pixelBufferPtr );
    if ( err != hipSuccess )
    {
      throw std::runtime_error( std::string( "RunConverterKernel failed: " ) + hipGetErrorString( err ) );
    }

    err = hipGraphicsUnmapResources( 1, &pboCudaResource );
    if ( err != hipSuccess )
    {
      throw std::runtime_error( std::string( "hipGraphicsUnmapResources failed: " ) + hipGetErrorString( err ) );
    }

    // notify view that we are done
    if ( mFinishedCallback != nullptr )
    {
      mFinishedCallback();
    }
  }
  catch ( const std::exception& /*e*/ )
  {
    // TODO: error handling
  }
  catch ( ... )
  {
    // TODO: error handling
  }
}

}
