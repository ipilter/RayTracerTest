#include "hip/hip_runtime.h"
#include <hip/hip_runtime.h>
#include <chrono>
#include <>

#include "RayTracerImpl.cuh"
#include "Random.cuh"
#include "DeviceUtils.cuh"
#include "Kernels.cuh"
#include "RenderBuffer.cuh"
#include "RaytracerCallback.h"

#include "Common\Logger.h"

namespace rt
{

RayTracerImpl::RayTracerImpl( const math::uvec2& imageSize
                              , const math::vec3& cameraPosition
                              , const math::vec2& cameraAngles
                              , const float fov
                              , const float focalLength
                              , const float aperture )
  : mBufferSize( imageSize )
  , mRenderBuffer( nullptr )
  , mSampleCountBuffer( nullptr )
  , mImageBuffer( nullptr )
  , mRandomStates( nullptr )
  , mCamera( new rt::ThinLensCamera( cameraPosition, cameraAngles, fov, focalLength, aperture ) )
  , mStopped( false )
  , mSceneTrianglesArray( 0 )
  , mSceneTrianglesTextureObject( 0 )
{
  try
  {
    random::CreateStates( mBufferSize, mRandomStates );
    rt::CreateRenderBuffer( mBufferSize, ChannelCount(), mRenderBuffer );
    rt::CreateSampleCountBuffer( mBufferSize, mSampleCountBuffer );
    rt::CreateImageBuffer( mBufferSize, mImageBuffer );

    logger::Logger::Instance() << "Raytracer created. Image buffer size: " << mBufferSize << "\n";
  }
  catch ( const std::exception& e )
  {
    logger::Logger::Instance() << "RayTracerImpl construction failed. Reason: " << e.what() << "\n";
  }
}

RayTracerImpl::~RayTracerImpl()
{
  // If rendering task is in progress, cancel it
  mStopped = true;
  if ( mThread.joinable() )
  {
    mThread.join();
  }

  // delete texture object
  hipError_t ret = hipDestroyTextureObject( mSceneTrianglesTextureObject );
  mSceneTrianglesTextureObject = 0;

  // delete device data
  ret = hipFreeArray( mSceneTrianglesArray );
  mSceneTrianglesArray = 0;

  // TODO free temporary image buffer - storage of converted image pixels (RGBA)
  ReleaseBuffers();
}

void RayTracerImpl::Trace( const uint32_t iterationCount
                           , const uint32_t samplesPerIteration
                           , const uint32_t updateInterval )
{
  // cancel previous operation, if any
  if ( mThread.joinable() )
  {
    mStopped = true;
    mThread.join();
    mStopped = false;
  }

  // Run rendering function async. TODO use pool instead of creating a new thread
  mThread = std::thread( std::bind( &RayTracerImpl::TraceFunct
                                    , this
                                    , iterationCount
                                    , samplesPerIteration
                                    , updateInterval ) );
}

void RayTracerImpl::Stop()
{
  mStopped = true;
}

void RayTracerImpl::Resize( const math::uvec2& size )
{
  ReleaseBuffers();

  mBufferSize = size;
  random::CreateStates( mBufferSize, mRandomStates );
  rt::CreateRenderBuffer( mBufferSize, ChannelCount(), mRenderBuffer );
  rt::CreateSampleCountBuffer( mBufferSize, mSampleCountBuffer );
  rt::CreateImageBuffer( mBufferSize, mImageBuffer );
}

void RayTracerImpl::SetCameraParameters( const float fov
                                         , const float focalLength
                                         , const float aperture )
{
  mCamera->Fov( fov );
  mCamera->FocalLength( focalLength );
  mCamera->Aperture( aperture );
}

void RayTracerImpl::RotateCamera( const math::vec2& angles )
{
  mCamera->Rotate( angles );
}

void RayTracerImpl::UploadScene( const std::vector<float4>& hostData )
{
  if ( hostData.size() < 3ull || hostData.size() % 3ull != 0ull )
  {
    logger::Logger::Instance() << "UploadScene got invalid triangle list. Size = " << hostData.size() << ", " << "\n";
    return;
  }

  hipError_t ret = hipSuccess;
  if ( mSceneTrianglesTextureObject != 0 )
  {
    // delete texture object
    ret = hipDestroyTextureObject( mSceneTrianglesTextureObject );
    mSceneTrianglesTextureObject = 0;

    // delete device data
    ret = hipFreeArray( mSceneTrianglesArray );
    mSceneTrianglesArray = 0;
  }

  mNumberOfTriangles = static_cast<uint32_t>( hostData.size() / 3 );

  // the scene triangles are stored in a 1D CUDA texture of float4 for memory alignment
  // store two edges instead of vertices
  // each triangle (v0,v1,v2) is stored as three float4s: v0, v1-v0, v2-v0 (one vertex and two edges connecting tho that edge)

  // prepare device memory and copy data
  {
    // format
    hipChannelFormatDesc channelDesc( hipCreateChannelDesc<float4>() );

    // allocate memory
    const size_t width = mNumberOfTriangles * 3;
    size_t arraySize = width * sizeof( float4 ); // bytes
    ret = hipMallocArray( &mSceneTrianglesArray, &channelDesc, arraySize );

    // copy 1D array to device
    const size_t spitch = width * sizeof(float4); // bytes of width
    ret = hipMemcpy2DToArray( mSceneTrianglesArray, 0, 0, &hostData.front(), spitch, arraySize, 1, hipMemcpyHostToDevice );
  }

  // create texture object
  {
    // specify resource parameters
    hipResourceDesc resDesc;
    memset( &resDesc, 0, sizeof( hipResourceDesc ) );
    resDesc.resType = hipResourceTypeArray;
    resDesc.res.array.array = mSceneTrianglesArray;

    // specify texture object parameters
    hipTextureDesc texDesc;
    memset( &texDesc, 0, sizeof( hipTextureDesc ) );
    texDesc.addressMode[0] = hipAddressModeClamp;
    texDesc.filterMode = hipFilterModePoint;
    texDesc.readMode = hipReadModeElementType;

    ret = hipCreateTextureObject( &mSceneTrianglesTextureObject, &resDesc, &texDesc, NULL );
  }
}

void RayTracerImpl::SetUpdateCallback( rt::CallBackFunction callback )
{
  mUpdateCallback = callback;
}

void RayTracerImpl::SetFinishedCallback( rt::CallBackFunction callback )
{
  mFinishedCallback = callback;
}

hipError_t RayTracerImpl::RunConverterKernel()
{
  const dim3 threadsPerBlock( 32, 32, 1 );
  const dim3 blocksPerGrid( static_cast<uint32_t>( glm::ceil( mBufferSize.x / static_cast<float>( threadsPerBlock.x ) ) )
                            , static_cast<uint32_t>( glm::ceil( mBufferSize.y / static_cast<float>( threadsPerBlock.y ) ) )
                            , 1 );

  ConverterKernel<<<blocksPerGrid, threadsPerBlock>>>( mBufferSize
                                                       , ChannelCount()
                                                       , mRenderBuffer
                                                       , mSampleCountBuffer
                                                       , mImageBuffer );
  return hipGetLastError();
}

hipError_t RayTracerImpl::RunTraceKernel( const uint32_t sampleCount )
{
  // TODO fast way, do better!
  const dim3 threadsPerBlock( 32, 32, 1 );
  const dim3 blocksPerGrid( static_cast<uint32_t>( glm::ceil( mBufferSize.x / static_cast<float>( threadsPerBlock.x ) ) )
                            , static_cast<uint32_t>( glm::ceil( mBufferSize.y / static_cast<float>( threadsPerBlock.y ) ) )
                            , 1 );

  hipEvent_t start, stop;
  hipEventCreate( &start );
  hipEventCreate( &stop );

  hipEventRecord( start, 0 );
  TraceKernel<<<blocksPerGrid, threadsPerBlock>>> ( mRenderBuffer
                                                    , mSampleCountBuffer
                                                    , mBufferSize
                                                    , ChannelCount()
                                                    , *mCamera
                                                    , sampleCount
                                                    , mRandomStates
                                                    , mSceneTrianglesTextureObject
                                                    , mNumberOfTriangles);

  hipEventRecord( stop, 0 );
  hipEventSynchronize( stop ); // TODO: make this switchable (on/off)

  float time = 0.0f;
  hipEventElapsedTime( &time, start, stop );

  return hipGetLastError();
}

__host__ void RayTracerImpl::TraceFunct( const uint32_t iterationCount
                                         , const uint32_t samplesPerIteration
                                         , const uint32_t updateInterval )
{
  try
  {
    rt::ClearRenderBuffer( mBufferSize, ChannelCount(), mRenderBuffer );
    rt::ClearSampleCountBuffer( mBufferSize, mSampleCountBuffer );

    hipError_t err = hipSuccess;
    for ( uint32_t i( 0 ); !mStopped && i < iterationCount; ++i )
    {
      // TODO: make kernel call cancellable if possible (imageine long runtimer here, cancel operation would wait for this call)
      err = RunTraceKernel( samplesPerIteration );
      if ( err != hipSuccess )
      {
        throw std::runtime_error( std::string( "RunTraceKernel failed: " ) + hipGetErrorString( err ) );
      }

      // check if update is needed
      if ( mUpdateCallback != nullptr && i > 0 && updateInterval > 0 && i % updateInterval == 0 )
      {
        // wait for the scheduled commands to be executed
        hipDeviceSynchronize();

        //std::vector<uint32_t> ha( mBufferSize.x * mBufferSize.y, 0 );
        //rt::CopyDeviceDataToHost( mSampleCountBuffer, &ha.front(), mBufferSize.x * mBufferSize.y );

        // run render -> image conversion
        err = RunConverterKernel();
        if ( err != hipSuccess )
        {
          throw std::runtime_error( std::string( "RunConverterKernel failed: " ) + hipGetErrorString( err ) );
        }

        // notify view to update the view's texture
        mUpdateCallback( mImageBuffer, mBufferSize.x * mBufferSize.y * sizeof( rt::Color ) );
      }

      using namespace std::chrono_literals;
      //std::this_thread::sleep_for( 500ms );
    }

    // early reaturn if cancel was called on us
    if ( mStopped )
    {
      // TODO: any UI update?
      return;
    }

    // wait for the scheduled commands to be executed
    hipDeviceSynchronize();

    //std::vector<uint32_t> ha( mBufferSize.x * mBufferSize.y, 0 );
    //rt::CopyDeviceDataToHost( mSampleCountBuffer, &ha.front(), mBufferSize.x * mBufferSize.y );
    //auto idx = ( mBufferSize.x * mBufferSize.y ) - 1;
    //auto elem = ha[0];

    // run render -> image conversion
    err = RunConverterKernel();
    if ( err != hipSuccess )
    {
      throw std::runtime_error( std::string( "RunConverterKernel failed: " ) + hipGetErrorString( err ) );
    }

    // notify view that we are done
    if ( mFinishedCallback != nullptr )
    {
      mFinishedCallback( mImageBuffer, mBufferSize.x * mBufferSize.y * sizeof( rt::Color ) );
    }
  }
  catch ( const std::exception& /*e*/ )
  {
    // TODO: error handling
  }
  catch ( ... )
  {
    // TODO: error handling
  }
}

void RayTracerImpl::ReleaseBuffers()
{
  hipError_t err = hipFree( mRandomStates );
  if ( err != hipSuccess )
  {
    logger::Logger::Instance() << "Error: hipFree failed freeing mRandomStates. (" << hipGetErrorString( err ) << "\n";
  }

  err = hipFree( mRenderBuffer );
  if ( err != hipSuccess )
  {
    logger::Logger::Instance() << "Error: hipFree failed freeing mRenderBuffer. (" << hipGetErrorString( err ) << "\n";
  }

  err = hipFree( mSampleCountBuffer );
  if ( err != hipSuccess )
  {
    logger::Logger::Instance() << "Error: hipFree failed freeing mSampleCountBuffer. (" << hipGetErrorString( err ) << "\n";
  }

  err = hipFree( mImageBuffer );
  if ( err != hipSuccess )
  {
    logger::Logger::Instance() << "Error: hipFree failed freeing mImageBuffer. (" << hipGetErrorString( err ) << "\n";
  }
}

}
